#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 32


typedef struct rgb{
    unsigned int r;
    unsigned int g;
    unsigned int b;
} rgb;

__global__ void denoise_kernel(rgb* input, rgb* output, int width, int height)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i > 0 && j > 0 && i < width - 1 && j < height - 1)
    {
        int idx = j * width + i;
        output[idx].r = (2 * input[idx].r + input[idx - 1].r + input[idx + 1].r + input[idx - width].r + input[idx - width+1].r +input[idx - width-1].r+ input[idx + width].r+input[idx + width+1].r+input[idx + width-1].r) / 10;
        output[idx].g = (2 * input[idx].g + input[idx - 1].g + input[idx + 1].g + input[idx - width].g +  input[idx - width+1].g +input[idx - width-1].g+ input[idx + width].g+input[idx + width+1].g+input[idx + width-1].g) / 10;
        output[idx].b = (2 * input[idx].b + input[idx - 1].b + input[idx + 1].b + input[idx - width].b +  input[idx - width+1].b +input[idx - width-1].b+ input[idx + width].b+input[idx + width+1].b+input[idx + width-1].b) / 10;
    }
}

__global__ void histogram_equalization_kernel(rgb* input, rgb* output, int width, int height) {

    // Get the index of the current thread
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    // Check if the thread is within the bounds of the image
    if (x < width && y < height) {
        // Compute the histogram
        int hist[256] = { 0 };
        for (int i = 0; i < width * height; i++) {
            int gray = (input[i].r + input[i].g + input[i].b) / 3;
            hist[gray]++;
        }

        // Compute the cumulative distribution function (CDF) of the histogram
        int cdf[256] = { 0 };
        for (int i = 0; i < 256; i++) {
            cdf[i] = (i > 0) ? cdf[i - 1] + hist[i] : hist[i];
        }

        // Normalize the CDF
        for (int i = 0; i < 256; i++) {
            cdf[i] = (cdf[i] * 255) / (width * height);
        }

        // Apply the transformation to the input pixel
        int gray = (input[idx].r + input[idx].g + input[idx].b) / 3;
        output[idx].r = cdf[gray];
        output[idx].g = cdf[gray];
        output[idx].b = cdf[gray];
    }
}

__global__ void contrastEnhancementLuminosity(rgb *input, rgb *output, int width, int height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i==width-1 && j==height-1)
        printf("Thread %d %d", i, j);

    if (i < width && j < height) {
        int index = j * width + i;
        float r = input[index].r;
        float g = input[index].g;
        float b = input[index].b;
        float lum = 0.2126f * r + 0.7152f * g + 0.0722f * b;
        output[index].r = fminf(fmaxf((r - lum) * 1.2f + lum, 0.0f), 255.0f);
        output[index].g = fminf(fmaxf((g - lum) * 1.2f + lum, 0.0f), 255.0f);
        output[index].b = fminf(fmaxf((b - lum) * 1.2f + lum, 0.0f), 255.0f);        
    }
}

__global__ void image_sharpening_kernel(rgb* input, rgb* output, int width, int height) {
    
    // Get the index of the current thread
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    // Check if the thread is within the bounds of the image
    if (x < width && y < height) {

        // Define the sharpening filter
       float kernel[3][3] = {
           {-1, -1, -1},
           {-1, 9, -1},
           {-1, -1, -1}
        };

        // Initialize the pixel values for the output image
        float r = 0;
        float g = 0;
        float b = 0;

        // Compute the sharpened pixel values
        for (int i = -1; i <= 1; i++) {
            for (int j = -1; j <= 1; j++) {
                int row = y + i;
                int col = x + j;

                // Make sure the pixel is within the bounds of the image
                if (row >= 0 && row < height && col >= 0 && col < width) {
                    int offset = row * width + col;
                    r += input[offset].r * kernel[i + 1][j + 1];
                    g += input[offset].g * kernel[i + 1][j + 1];
                    b += input[offset].b * kernel[i + 1][j + 1];
                }
            }
        }

        // Normalize the pixel values and write to output
        output[idx].r = fminf(fmaxf(r, 0), 255);
        output[idx].g = fminf(fmaxf(g, 0), 255);
        output[idx].b = fminf(fmaxf(b, 0), 255);
    }
}

int main() {

    FILE* input_file = fopen("input.txt", "r");

    // first line contains width and height
    int width, height;

    fscanf(input_file, "%d %d\n", &width, &height);
    
    printf("%d %d\n", width, height);

    // remaining lines contain rgb values
    rgb *input = (rgb*)malloc(width * height * sizeof(rgb));

    for (int i = 0; i < width * height; i++) {
        fscanf(input_file, "%d %d %d\n", &input[i].r, &input[i].g, &input[i].b);
    }

    fclose(input_file);

    rgb *d_input;
    hipMalloc((void**)&d_input, width * height * sizeof(rgb));
    hipMemcpy(d_input, input, width * height * sizeof(rgb), hipMemcpyHostToDevice);

    rgb *d_output;
    hipMalloc((void**)&d_output, width * height * sizeof(rgb));

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);

    //denoise_kernel<<<dimGrid, dimBlock>>>(d_input, d_output, width, height);
    histogram_equalization_kernel<<<dimGrid, dimBlock>>>(d_input, d_output, width, height);
    //image_sharpening_kernel<<<dimGrid, dimBlock>>>(d_input, d_output, width, height);
    //contrastEnhancementLuminosity<<<dimGrid, dimBlock>>>(d_input, d_output, width, height);


    rgb *output = (rgb*)malloc(width * height * sizeof(rgb) );
    hipMemcpy(output, d_output, width * height * sizeof(rgb), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    FILE* output_file = fopen("output.txt", "w");

    // write width and height 
    fprintf(output_file, "%d %d\n", width, height);

    for (int i = 0; i < width * height; i++) {
        fprintf(output_file, "%d %d %d\n", output[i].r, output[i].g, output[i].b);
    }

    fclose(output_file);

    free(input);
    free(output);

    return 0;
}